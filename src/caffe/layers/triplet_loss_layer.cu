#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void TripletLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // p
      diff_ap_.mutable_gpu_data());  // a_i-p_i
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[2]->gpu_data(),  // n
      diff_an_.mutable_gpu_data());  // a_i-n_i
  caffe_gpu_sub(
      count,
      bottom[1]->gpu_data(),  // p
      bottom[2]->gpu_data(),  // n
      diff_pn_.mutable_gpu_data());  // p_i-n_i
  caffe_gpu_powx(
      count,
      diff_ap_.mutable_gpu_data(),  // a_i-p_i
      Dtype(2),
      diff_sq_ap_.mutable_gpu_data());  // (a_i-p_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_ap_.gpu_data(),  // (a_i-p_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      dist_sq_ap_.mutable_gpu_data());  // \Sum (a_i-p_i)^2
  caffe_gpu_powx(
      count,
      diff_an_.mutable_gpu_data(),  // a_i-n_i
      Dtype(2),
      diff_sq_an_.mutable_gpu_data());  // (a_i-n_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_an_.gpu_data(),  // (a_i-n_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      dist_sq_an_.mutable_gpu_data());  // \Sum (a_i-n_i)^2
  Dtype margin = this->layer_param_.triplet_loss_param().margin();
  Dtype loss(0.0);
  const Dtype* sample = bottom[3]->cpu_data();
  for (int i = 0; i < bottom[0]->num(); ++i) {
    loss += sample[i] * std::max(margin + dist_sq_ap_.cpu_data()[i] - dist_sq_an_.cpu_data()[i], Dtype(0.0));
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLForward(const int count, const int channels,
    const Dtype margin, const Dtype alpha, const Dtype* sample,
    const Dtype* diff, const Dtype* dist_sq_ap_, const Dtype* dist_sq_an_,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    Dtype mdist(0.0);
    mdist = margin + dist_sq_ap_[n] - dist_sq_an_[n];
    if (mdist > 0.0) {
      bottom_diff[i] = alpha * sample[n] * diff[i];
    } else {
      bottom_diff[i] = 0;
    }
  }
}

template <typename Dtype>
void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) { 
  const int count = bottom[0]->count();
  const int channels = bottom[0]->channels();
  Dtype margin = this->layer_param_.triplet_loss_param().margin();
  for (int i = 0; i < 3; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i < 2) ? -1 : 1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      if (i == 0) {
        CLLForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, channels, margin, alpha,
            bottom[3]->gpu_data(),
            diff_pn_.gpu_data(),  // the cached eltwise difference between p and n
            dist_sq_ap_.gpu_data(),  // the cached square distance between a and p
            dist_sq_an_.gpu_data(), // the cached square distance between a and n
            bottom[i]->mutable_gpu_diff());
        CUDA_POST_KERNEL_CHECK;
      } else if (i == 1) {
        CLLForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, channels, margin, alpha,
            bottom[3]->gpu_data(),
            diff_ap_.gpu_data(),  // the cached eltwise difference between a and p
            dist_sq_ap_.gpu_data(),  // the cached square distance between a and p
            dist_sq_an_.gpu_data(), // the cached square distance between a and n
            bottom[i]->mutable_gpu_diff());
        CUDA_POST_KERNEL_CHECK;
      } else if (i == 2) {
        CLLForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, channels, margin, alpha,
            bottom[3]->gpu_data(),
            diff_an_.gpu_data(),  // the cached eltwise difference between a and n
            dist_sq_ap_.gpu_data(),  // the cached square distance between a and p
            dist_sq_an_.gpu_data(),  // the cached square distance between a and n
            bottom[i]->mutable_gpu_diff());
        CUDA_POST_KERNEL_CHECK;
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);

}  // namespace caffe
